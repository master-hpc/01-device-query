//
// TP: Exploration de la machine
// Compl�ter les TODOs
//


#include <hip/hip_runtime.h>
#include<iostream>

int main (int argc, char ** argv) {
    
    // Nombre de GPU sur la machine supporant CUDA
    int devices_count = 0;
    hipGetDeviceCount(&devices_count);
    
    std::cout << "Cette machine est équiée de " << devices_count << " GPU(s) supportant CUDA" << std::endl;
    // Lire les propri�t�s de chaque GPU
    for (int device_index = 0; device_index < devices_count; ++device_index) {
        
        std::cout << "============" << std::endl << "GPU index: " << device_index << std::endl;
        hipDeviceProp_t device_properties;
        hipGetDeviceProperties(&device_properties, device_index);
        
        std::cout << "Nom du GPU: " << device_properties.name << std::endl;
        std::cout << "Compute Capability: " << device_properties.major << "." << device_properties.minor << std::endl;
        std::cout << "Nombre de SMs: " << device_properties.multiProcessorCount << std::endl;
        std::cout << "Taille du warp: " << device_properties.warpSize << std::endl;

        // TODO: Imprimer d'autres propri�t�s
        // TIP: Voir https://stackoverflow.com/a/32531982/3503855        
    }
    
    // Consulter le lien suivant pour plus d'information sur la structure "cudaDeviceProp"
    // http://developer.download.nvidia.com/compute/cuda/3_2_prod/toolkit/docs/online/structcudaDeviceProp.html

    return 0;
}
